#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <GLFW/glfw3.h>
#include<windows.h> 

struct Body {
    float3 position;
    float3 velocity;
    float mass;
};

#define G 6.67e-11
#define epsilon 1e-11
const int width = 800;
const int height = 600;
const int NUMBODIES = 1000;

int generateSeed() {
    return static_cast<int>(time(NULL));
}

__global__ 
void noramlize(Body* arr,Body *dest) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < NUMBODIES; i += stride) {
        // Normalize position coordinates to NDC range [-1, 1]
        dest[i].position.x = arr[i].position.x / 1.0e11; // Divide by max value to get [-1, 1] range
        dest[i].position.y = arr[i].position.y / 1.0e11;
        dest[i].position.z = arr[i].position.z / 1.0e11;
    }
}

__global__
void initialize(Body* deviceBodies, int num_bodies, int seed) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < num_bodies; i += stride) {

        hiprandState state;
        hiprand_init(seed, i, 0, &state); // seed, sequence, offset, state

        deviceBodies[i].position.x = hiprand_uniform(&state) * 2.0e11 - 1.0e11; // Range: -1.0e11 to 1.0e11 meters
        deviceBodies[i].position.y = hiprand_uniform(&state) * 2.0e11 - 1.0e11; // Range: -1.0e11 to 1.0e11 meters
        deviceBodies[i].position.z = hiprand_uniform(&state) * 2.0e11 - 1.0e11; // Range: -1.0e11 to 1.0e11 meters


        deviceBodies[i].velocity.x = hiprand_uniform(&state) * (100000.0 - 15000.0) + 15000.0; // Range: 15,000 m/s to 100,000 m/s
        deviceBodies[i].velocity.y = hiprand_uniform(&state) * (100000.0 - 15000.0) + 15000.0; // Range: 15,000 m/s to 100,000 m/s
        deviceBodies[i].velocity.z = hiprand_uniform(&state) * (100000.0 - 15000.0) + 15000.0; // Range: 15,000 m/s to 100,000 m/s



        deviceBodies[i].mass = hiprand_uniform(&state) * (1.989e30 - 3.3011e23) + 3.3011e23;


    }
}



__global__ void computeAccn(Body* bodies, int num_bodies, float3* accelerations, float dt) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < num_bodies; i += stride) {
        float3 totalforce = { 0.0f,0.0f,0.0f };
        for (int j = 0; j < num_bodies; j++) {
            if (j != i) {
                float dist_x = bodies[j].position.x - bodies[i].position.x;
                float dist_y = bodies[j].position.y - bodies[i].position.y;
                float dist_z = bodies[j].position.z - bodies[i].position.z;

                float3 dist_vector = make_float3(dist_x, dist_y, dist_z);
                float dist_sq = dist_vector.x * dist_vector.x + dist_vector.y * dist_vector.y + dist_vector.z * dist_vector.z;
                float dist_cubed = sqrt(dist_sq) * sqrt(dist_sq) * sqrt(dist_sq);


                /*
                  Fij=(Gm1m2/(r^2+e^2)^3/2)rij

                */

                //printf("mass1 %f mass2 %f\n",bodies[i].mass,bodies[j].mass);
                float force_x = ((G * bodies[i].mass * bodies[j].mass) / dist_cubed) * dist_x;
                //printf("Distance %f\n", dist_x);
                //printf("Force %f\n",force_x);
                float force_y = G * bodies[i].mass * bodies[j].mass / dist_cubed * dist_vector.y;
                float force_z = G * bodies[i].mass * bodies[j].mass / dist_cubed * dist_vector.z;

                totalforce.x += force_x;
                totalforce.y += force_y;
                totalforce.z += force_z;


                //printf("Body %d DistCubed: %f\n", i, dist_cubed);
                //printf("Body %d Force: (%f, %f, %f)\n", i, totalforce.x, totalforce.y, totalforce.z);



            }

        }



        accelerations[i].x = totalforce.x / bodies[i].mass;
        accelerations[i].y = totalforce.y / bodies[i].mass;
        accelerations[i].z = totalforce.z / bodies[i].mass;
        //printf("Body %d Acceleration: (%f, %f, %f)\n", i, accelerations[i].x, accelerations[i].y, accelerations[i].z);
        bodies[i].velocity.x += accelerations[i].x * dt; //integral technically
        bodies[i].velocity.y += accelerations[i].y * dt;
        bodies[i].velocity.z += accelerations[i].z * dt;

        bodies[i].position.x += bodies[i].velocity.x * dt;
        bodies[i].position.y += bodies[i].velocity.y * dt;
        bodies[i].position.z += bodies[i].velocity.z * dt;
    }
}



// Function to display OpenGL scene
void display(GLFWwindow* window, Body* points, int numPoints) {
    glClear(GL_COLOR_BUFFER_BIT);
    glColor3f(1.0, 1.0, 1.0); // White color

    // Draw points
    glBegin(GL_POINTS);
    for (int i = 0; i < numPoints; i++) {
        glVertex2f(points[i].position.x, points[i].position.y);
    }
    glEnd();

    glfwSwapBuffers(window);
}

int main() {
    // Initialize GLFW
    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return -1;
    }

    // Create a windowed mode window and its OpenGL context
    GLFWwindow* window = glfwCreateWindow(width, height, "CUDA GLFW OpenGL Random Points", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }

    // Make the window's context current
    glfwMakeContextCurrent(window);

    // Initialize CUDA
    hipSetDevice(0);

    



    int threads_per_block = 256;
    int blocks_per_grid = (NUMBODIES - 1 + threads_per_block) / threads_per_block;
    int seed = generateSeed();
    Body* deviceBodies;
    hipMalloc((void**)&deviceBodies, NUMBODIES * sizeof(Body));

    initialize <<< blocks_per_grid, threads_per_block >>> (deviceBodies, NUMBODIES, seed);
    hipDeviceSynchronize();

    // Allocate memory for points on the host
    Body* points = new Body[NUMBODIES * 2];

    // Copy points from device to host
    hipMemcpy(points, deviceBodies, sizeof(float) * NUMBODIES * 2, hipMemcpyDeviceToHost);

    Body* normalized_bodies = (Body*)malloc(sizeof(Body) * NUMBODIES);
    Body* d_norm_bodies;
    hipMalloc((void**)&d_norm_bodies, NUMBODIES * sizeof(Body));

    noramlize << <blocks_per_grid, threads_per_block >> > (deviceBodies,d_norm_bodies);

    hipMemcpy(normalized_bodies, d_norm_bodies, sizeof(float) * NUMBODIES * 2, hipMemcpyDeviceToHost);


    float3* deviceAccelerations;


    float3* hostAccelerations = (float3*)malloc(sizeof(float3) * NUMBODIES);

    hipMalloc((void**)&deviceAccelerations, NUMBODIES * sizeof(float3));


    float dt = 1000;
    int frame_cnt = 0;

    // Loop until the user closes the window
    while (!glfwWindowShouldClose(window)) {
        // Render here
        display(window, normalized_bodies, NUMBODIES);
        computeAccn <<< blocks_per_grid, threads_per_block >>> (deviceBodies, NUMBODIES, deviceAccelerations, dt);

        hipMemcpy(points, deviceBodies, NUMBODIES, hipMemcpyDeviceToHost);
        noramlize << <blocks_per_grid, threads_per_block >> > (deviceBodies, d_norm_bodies);
        hipMemcpy(normalized_bodies, d_norm_bodies, sizeof(float) * NUMBODIES * 2, hipMemcpyDeviceToHost);

        //generateRandomPoints << <(numPoints + 255) / 256, 256 >> > (d_points, time(NULL));
        //hipMemcpy(points, d_points, sizeof(float) * numPoints * 2, hipMemcpyDeviceToHost);

        //Sleep(500);
        // Poll for and process events
        std::cout << "Frame no= " << frame_cnt++ << std:: endl;
        glfwPollEvents();
        
    }

    // Free memory
    delete[] points;
    hipFree(deviceBodies);
    hipFree(d_norm_bodies);
    free(normalized_bodies);
    
    // Terminate GLFW
    glfwTerminate();

    return 0;
}